#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "GamePlanner.cuh"
#include "Smoothing.cuh"
#include <stdio.h>
#include <iostream>
using namespace std;

extern "C" __declspec(dllexport) void __cdecl Start(float, float, float, int, int, int, float *, float*, float*, int, int, float, float);
extern "C" __declspec(dllexport) void __cdecl GPlannerBallState(float*, int, int, float*, float*, float*, float*, float*, int*);
extern "C" __declspec(dllexport) void __cdecl GPlannerScore(float*, int, float*, float*, float*);
extern "C" __declspec(dllexport) float __cdecl ForceTree(float*, int*, int, float*, float*, int, float*, int, float, float, int);
extern "C" __declspec(dllexport) void __cdecl ShutDown();



extern void Start(float maxRobotAccel, float maxRobotSpeed, float ballDecel, int maxPathCount, int maxRRTCount, int maxRobotCount, float * DataScoreX, float* DataScoreY, float* eachRegionCount, int RegionCount, int maxSampleCount, float sigmaX, float sigmaY)
{
	hipSetDevice(0);
	hipDeviceReset();
	//hipCtx_t cuContext;
	int  device = 0;
	hipGetDevice(&device);
	/*hipCtxCreate(&cuContext, 0, device);
	hipMemGetInfo(&memory_free, &memory_total);
	cout << "freeMem: " << memory_free << "    totalMem: " << memory_total;
	cuCtxDetach(cuContext);*/
	GamePlannerInit(maxRobotAccel, maxRobotSpeed, ballDecel, DataScoreX, DataScoreY, eachRegionCount, RegionCount, maxSampleCount, sigmaX, sigmaY);
	//ElasticInit(maxPathCount, maxRRTCount);
	/*hipMemGetInfo(&memory_free, &memory_total);
	cout << "freeMem: " << memory_free << "    totalMem: " << memory_total;*/
}
extern void GPlannerBallState(float* robots, int RobotCounts, int N, float* ball, float* Heads, float* Tails, float* TimeHeads, float* TimeTails, int* histo)
{

	GObjectState* robotStates = new GObjectState[RobotCounts];
	for (int i = 0; i < RobotCounts; i++)
	{
		robotStates[i].Location.X = robots[4 * i];
		robotStates[i].Location.Y = robots[4 * i + 1];
		robotStates[i].Speed.X = robots[4 * i + 2];
		robotStates[i].Speed.Y = robots[4 * i + 3];
		//cout<< "Robot" << i << ": " << robotStates[i].Location.X << "            " << robotStates[i].Location.Y  << "            "<< robotStates[i].Speed.X  << "            " << robotStates[i].Speed.Y  << "\n";
	}

	GObjectState BallState;
	BallState.Location.X = ball[0];
	BallState.Location.Y = ball[1];
	BallState.Speed.X = ball[2];
	BallState.Speed.Y = ball[3];

	CalculateBallState(robotStates, RobotCounts, N, BallState, Heads, Tails, TimeHeads, TimeTails, histo);
}
extern void GPlannerScore(float* Robots, int RobotCount, float* Phi, float* Kdx, float* Kdy)
{
	if (_RegionCount > 0 && _maxSampleCount > 0 && RobotCount)
	{
		dim3 Block(_RegionCount, _maxSampleCount);
		dim3 Grid(RobotCount);

		error = hipMemcpy(DevRobots, Robots, RobotCount * sizeof(float), hipMemcpyHostToDevice);
		//cout << "memcpyRobotsX: "<< error << "\n";
		error = hipMemcpy(DevRobots + maxRobotCount, Robots + maxRobotCount, RobotCount * sizeof(float), hipMemcpyHostToDevice);
		//cout << "memcpyRobotsY: "<< error << "\n";
		GaussianKernel << <Grid, Block >> >(DevDataX, _RegionCount * _maxSampleCount, DevDataY, _RegionCount * _maxSampleCount, DevEachDataCount, _RegionCount, _RegionCount, DevRobots, 2 * maxRobotCount, RobotCount, _sigmaX, _sigmaY, _maxSampleCount, DevPhi, _maxSampleCount * _RegionCount * maxRobotCount, DevKdx, _maxSampleCount * _RegionCount * maxRobotCount, DevKdy, _maxSampleCount * _RegionCount * maxRobotCount);
		//error = hipGetLastError();
		//cout << "GaussianKernel: "<< error << "\n";
		error = hipMemcpy(Phi, DevPhi, RobotCount *_RegionCount * _maxSampleCount * sizeof(float), hipMemcpyDeviceToHost);
		//cout << "memcpyPhi: "<< error << "\n";
		error = hipMemcpy(Kdx, DevKdx, RobotCount *_RegionCount * _maxSampleCount * sizeof(float), hipMemcpyDeviceToHost);
		//cout << "memcpyKdx: "<< error << "\n";
		error = hipMemcpy(Kdy, DevKdy, RobotCount *_RegionCount * _maxSampleCount * sizeof(float), hipMemcpyDeviceToHost);
		//cout << "memcpyKdy: "<< error << "\n";
	}
}
extern float ForceTree(float* Path, int* eachPathCount, int RobotCount, float* avoid, float* finalPath, int SmoothingCount, float* Obstacles, int ObstacleCount, float Kspring, float Kspring2, int n)
{
	_kSpring = Kspring;
	_kSpring2 = Kspring2;
	N = n;
	hipError_t error;
	int maxP = -MaxPathCount;

	if (RobotCount > 0)
	{
		for (int i = 0; i < RobotCount; i++)
		{
			error = hipMemcpy2D(DevPath + i * PathPitch, PathPitch * sizeof(float), Path + i * 2 * MaxPathCount, 2 * MaxPathCount * sizeof(float), 2 * eachPathCount[i] * sizeof(float), 1, hipMemcpyHostToDevice);
			//cout << "PathMemCpy for Robot" << i << ": "<< error << "\n";
			if (eachPathCount[i] > maxP)
				maxP = eachPathCount[i];
		}
		//	hipDeviceSynchronize();
		error = hipMemcpy2D(DevObs, ObsPitch * sizeof(float), Obstacles, ObstacleCount * sizeof(float), ObstacleCount * sizeof(float), 2, hipMemcpyHostToDevice);
		//	cout << "ObsMemcpy: "<< error << "\n";
		error = hipMemcpy(DevEachPathCount, eachPathCount, RobotCount * sizeof(int), hipMemcpyHostToDevice);
		//	cout << "EachPathMemcpy: "<< error << "\n";
		error = hipMemcpy2D(DevAvoid, AvoidPitch * sizeof(float), avoid, RobotCount * sizeof(float), RobotCount * sizeof(float), 4, hipMemcpyHostToDevice);
		//	cout << "avoidMemcpy: "<< error << "\n";

		dim3 Block(128, 2, 1);
		dim3 Grid((maxP + Block.x - 1) / Block.x, RobotCount, 1);

		dim3 Block2(256, 1, 1);
		dim3 Grid2((maxP + Block2.x - 1) / Block2.x, /*ObstacleCount * */RobotCount, 1);
		/*hipStreamSynchronize(streams[RobotCount + 1]);
		hipStreamSynchronize(streams[RobotCount + 2]);
		hipStreamSynchronize(streams[RobotCount]);*/

		//for(int i = 0; i < RobotCount; i ++)
		//{
		//	hipStreamSynchronize(streams[i]);
		//}
		for (int i = 0; i < SmoothingCount; i++)
		{
			CalculateForcesKernel << <Grid, Block >> >(DevForce, DevEachPathCount, RobotCount, ForcePitch, _kSpring, _kSpring2, N);
			//	error = hipGetLastError();
			//	cout << "CalculateForceKernell iter " << i << ": " << error << "\n";
			ReCalculatePath << <Grid2, Block2 >> >(DevPath, DevEachPathCount, RobotCount, PathPitch, ObstacleCount);
			//	error = hipGetLastError();
			//	cout << "ReCalcPathKernell iter " << i << ": " << error << "\n";
		}
		//hipMemcpy2D(finalPath, PathCount * sizeof(float), DevPath, PathPitch * sizeof(float), PathCount * sizeof(float), 2, hipMemcpyDeviceToHost);
		for (int i = 0; i < RobotCount; i++)
		{
			error = hipMemcpy2D(finalPath + i * 2 * MaxPathCount, 2 * MaxPathCount * sizeof(float), DevPath + i * PathPitch, PathPitch * sizeof(float), 2 * eachPathCount[i] * sizeof(float), 1, hipMemcpyDeviceToHost);
			//		cout << "finalPathMemCpy for Robot" << i << ": "<< error << "\n";
		}
		//
		//	error = hipDeviceSynchronize();
		//	cout << "Sync: " << error << "\n";
	}
	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//float time;
	//hipEventElapsedTime(&time, start, stop);
	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	return 1;
}
extern void ShutDown()
{
	DisposeGamePlanner();
	//DisposeElastic();
}



__global__ void GaussianKernel(float* DataX, int DataXLen0, float* DataY, int DataYLen0, float* EachDataCount, int EachDataCountLen0, int RegionCount, float* opps, int oppsLen0, int RobotCount, float sigmax, float sigmay, int maxCount, float* DevPhi, int DevPhiLen0, float* DevKdx, int DevKdxLen0, float* DevKdy, int DevKdyLen0)
{
	int x = blockIdx.x;
	int x2 = threadIdx.x;
	int y = threadIdx.y;
	if (x < RobotCount && x2 < RegionCount && (float)y < EachDataCount[(x2)])
	{
		float num = DataX[(x2 * maxCount + y)];
		float num2 = DataY[(x2 * maxCount + y)];
		float num3 = expf(-(opps[(x)] - num) * (opps[(x)] - num) / sigmax - (opps[(x + 6)] - num2) * (opps[(x + 6)] - num2) / sigmay);
		DevKdx[(x * RegionCount * maxCount + x2 * maxCount + y)] = -2 * ((opps[(x)] - num) / sigmax) * num3;
		DevKdy[(x * RegionCount * maxCount + x2 * maxCount + y)] = -2 * ((opps[(x + 6)] - num2) / sigmay) * num3;
		DevPhi[(x * RegionCount * maxCount + x2 * maxCount + y)] = num3;
	}
}
__global__ void MultiObjectKernel(GObjectState Ball, float maxBallDeccel, float* RobotTimes, int RobotTimesLen0, int RobotTimesLen1, int N, GPosition2D* Heads, int HeadsLen0, int HeadsLen1, GPosition2D* Tails, int TailsLen0, int TailsLen1, float* TimesHeads, int TimesHeadsLen0, int TimesHeadsLen1, float* TimesTails, int TimesTailsLen0, int TimesTailsLen1, int* histo, int histoLen0, int histoLen1)
{
	int num = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x = threadIdx.x;
	bool flag = false;
	bool flag2 = false;
	bool flag3 = false;
	__shared__ bool sharedarray[257];

	int arrayLen0 = 257;
	GVector2D gVector2D = GVector2D(0, 0);
	if (num < N)
	{
		float num2 = (float)(num * 10) / 100;
		float num3 = Ball.Speed.Size();
		float num4 = num3 / maxBallDeccel;
		float num5 = num3 * num3 / (2 * maxBallDeccel);
		GPosition2D gPosition2D = Ball.Location.Add(Ball.Speed.GetNormalizeToCopy(num2));
		float num6 = RobotTimes[(y)* RobotTimesLen1 + (num)];
		if (fabsf(num2 - num5) <= 0.1)
		{
			flag = true;
		}
		else
		{
			float num7 = num3 * num3 - 2 * maxBallDeccel * num2;
			if (num7 >= 0)
			{
				float num8 = (sqrtf(num7) - num3) / -maxBallDeccel;
				flag = (num6 <= num8);
			}
			else
			{
				flag = false;
			}
		}
		if (x == 0 && flag)
		{
			flag2 = true;
		}
		sharedarray[(x)] = flag;
		if (x == 255 || num == N - 1)
		{
			sharedarray[(x + 1)] = false;
		}
		__syncthreads();
		bool flag4 = sharedarray[(x + 1)];
		__syncthreads();
		if (flag && !flag4)
		{
			flag3 = true;
		}
		if (!flag && flag4)
		{
			flag2 = true;
		}
		if (flag2)
		{
			int num9 = atomicAdd(&histo[(y)* histoLen1 + (0)], 1);

			if (num9 < 4)
			{
				Heads[(y)* HeadsLen1 + (num9)] = gPosition2D;

				TimesHeads[(y)* TimesHeadsLen1 + (num9)] = num6;
			}
		}
		if (flag3)
		{
			int num9 = atomicAdd(&histo[(y)* histoLen1 + (1)], 1);
			if (num9 < 4)
			{
				Tails[(y)* TailsLen1 + (num9)] = gPosition2D;
				TimesTails[(y)* TimesTailsLen1 + (num9)] = num6;
			}
		}
	}
}
__global__ void ClaculateTimeKernel(GObjectState Ball, GObjectState* States, int StatesLen0, int N, float maxRobotSpeed, float maxBallDeccel, float maxRobotAccel, float* MinTime, int MinTimeLen0, int MinTimeLen1)
{
	__shared__ float array[8 * 32];

	int arrayLen0 = 8;
	int arrayLen1 = 32;
	int num = blockIdx.x * blockDim.x + threadIdx.x;
	GVector2D gVector2D = GVector2D(0, 0);
	if (num < N)
	{
		int i = blockDim.y;
		float a_max = fmaxf(0.01, maxRobotAccel * cosf((float)threadIdx.y * 3.141593 / 62));
		float a_max2 = fmaxf(0.01, maxRobotAccel * sinf((float)threadIdx.y * 3.141593 / 62));
		gVector2D = Ball.Location.Add(Ball.Speed.GetNormalizeToCopy((float)(num * 10) / 100)).Sub(States[(blockIdx.y)].Location);
		if (gVector2D.Size() >= 0.09)
		{
			gVector2D.NormalizeTo(gVector2D.Size() - 0.09);
		}
		else
		{
			gVector2D.X = (gVector2D.Y = 0);
		}
		float num2 = CalculateTime(gVector2D.X, States[(blockIdx.y)].Speed.X, maxRobotSpeed, a_max);
		float num3 = CalculateTime(gVector2D.Y, States[(blockIdx.y)].Speed.Y, maxRobotSpeed, a_max2);
		array[(threadIdx.x) * arrayLen1 + (threadIdx.y)] = ((num2 > num3) ? num2 : num3);
		__syncthreads();
		while (i > 1)
		{
			i >>= 1;
			if (threadIdx.y < i)
			{
				if (array[(threadIdx.x) * arrayLen1 + (threadIdx.y + i)] < array[(threadIdx.x) * arrayLen1 + (threadIdx.y)])
				{
					array[(threadIdx.x) * arrayLen1 + (threadIdx.y)] = array[(threadIdx.x) * arrayLen1 + (threadIdx.y + i)];
				}
			}
			__syncthreads();
		}
		__syncthreads();
		if (threadIdx.y == 0)
		{
			MinTime[(blockIdx.y) * MinTimeLen1 + (num)] = array[(threadIdx.x) * arrayLen1 + (0)];
		}
	}
}
__device__ float CalculateTime(float dR, float v0, float v_max, float a_max)
{
	float num = 0;
	float num2 = sign(dR);
	float result;
	if (dR == 0)
	{
		if (fabsf(v0) <= 0.02)
		{
			result = 0;
			return result;
		}
		float num3 = a_max * -sign(v0);
		float num4 = -v0 / num3;
		float num5 = -v0 * v0 / (2 * num3);
		dR -= num5;
		v0 = 0;
		float num6 = v_max / a_max;
		float value = v_max * v_max / num3;
		float num7;
		if (fabsf(value) <= fabsf(dR))
		{
			num7 = 2 * num6 + (fabsf(dR) - fabsf(value)) / v_max;
		}
		else
		{
			num7 = 2 * sqrtf(num3 * dR) / a_max;
		}
		num = num4 + num7;
	}
	else
	{
		float num8 = sign(v0 * dR);
		if (num8 < 0)
		{
			float num3 = a_max * sign(dR);
			float num4 = -v0 / num3;
			float num5 = -v0 * v0 / (2 * num3);
			dR -= num5;
			v0 = 0;
			float num6 = v_max / a_max;
			float value = v_max * v_max / num3;
			float num7;
			if (fabsf(value) <= fabsf(dR))
			{
				num7 = 2 * num6 + (fabsf(dR) - fabsf(value)) / v_max;
			}
			else
			{
				num7 = 2 * sqrtf(num3 * dR) / a_max;
			}
			num = num4 + num7;
		}
		else
		{
			float num3 = -sign(dR) * a_max;
			float num4 = -v0 / num3;
			float num5 = -v0 * v0 / (2 * num3);
			if (fabsf(num5) > fabsf(dR))
			{
				dR -= num5;
				v0 = 0;
				num3 = sign(dR) * a_max;
				float num6 = v_max / a_max;
				float value = v_max * v_max / num3;
				float num7;
				if (fabsf(value) <= fabsf(dR))
				{
					num7 = 2 * num6 + (fabsf(dR) - fabsf(value)) / v_max;
				}
				else
				{
					num7 = 2 * sqrtf(num3 * dR) / a_max;
				}
				num = num7 + num4;
			}
			else
			{
				num3 = sign(dR) * a_max;
				float num9 = sign(dR) * v_max;
				float value = (2 * num9 * num9 - v0 * v0) / (2 * num3);
				float num6 = (2 * num9 - v0) / num3;
				float num7;
				if (fabsf(value) <= fabsf(dR))
				{
					num7 = num6 + (fabsf(dR) - fabsf(value)) / v_max;
				}
				else
				{
					float num10 = sign(dR) * sqrtf(num3 * dR + v0 * v0 / 2);
					num7 = (2 * num10 - v0) / num3;
				}
				num = num7;
			}
		}
	}
	result = num;
	return result;
}
__device__ float sign(float x)
{
	return (x == 0) ? 0 : (x / fabsf(x));
}

void GamePlannerInit(float maxRobotAccel, float maxRobotSpeed, float ballDecel, float* DataScoreX, float* DataScoreY, float* eacheRegionCount, int RegionCount, int maxSampleCount, float sigmaX, float sigmaY)
{
	_maxRobotAccel = maxRobotAccel;
	_maxRobotSpeed = maxRobotSpeed;
	_ballDecel = ballDecel;

	pHeads = new GPosition2D[2 * maxRobotCount * maxLines];
	pTails = new GPosition2D[2 * maxRobotCount * maxLines];

	error = hipMalloc((void**)&devHeads2D, 2 * maxRobotCount * sizeof(GPosition2D) * maxLines);
	cout << "AllocHeads: " << error << "\n";
	error = hipMalloc((void**)&devTails2D, 2 * maxRobotCount * sizeof(GPosition2D) * maxLines);
	cout << "AllocTails: " << error << "\n";
	error = hipMalloc((void**)&devHisto2D, 2 * maxRobotCount * sizeof(int) * 2);
	cout << "AllocHisto: " << error << "\n";
	error = hipMalloc((void**)&devtimehead2D, 2 * maxRobotCount * sizeof(float) * maxLines);
	cout << "AlloctimeHead: " << error << "\n";
	error = hipMalloc((void**)&devtimetail2D, 2 * maxRobotCount * sizeof(float) * maxLines);
	cout << "AlloctimeTail: " << error << "\n";
	error = hipMalloc((void**)&devMinTime2D, 2 * maxRobotCount * sizeof(float) * maxPoints);
	cout << "AllocMintime: " << error << "\n";
	error = hipMalloc((void**)&states, 2 * maxRobotCount * sizeof(GObjectState));
	cout << "AllocStates: " << error << "\n";
	zeroArray = new int[2 * maxRobotCount * 2];
	for (int i = 0; i < 2 * maxRobotCount * 2; i++)
		zeroArray[i] = 0;

	_sigmaX = sigmaX;
	_sigmaY = sigmaY;
	_RegionCount = RegionCount;
	_maxSampleCount = maxSampleCount;
	error = hipMalloc((void**)&DevDataX, RegionCount * maxSampleCount * sizeof(float));
	cout << "AllocDataX: " << error << "\n";
	error = hipMalloc((void**)&DevDataY, RegionCount * maxSampleCount * sizeof(float));
	cout << "AllocDataY: " << error << "\n";
	error = hipMalloc((void**)&DevPhi, RegionCount * maxSampleCount * maxRobotCount * sizeof(float));
	cout << "AllocPhi: " << error << "\n";
	error = hipMalloc((void**)&DevKdx, RegionCount * maxSampleCount * maxRobotCount * sizeof(float));
	cout << "AllocKdx: " << error << "\n";
	error = hipMalloc((void**)&DevKdy, RegionCount * maxSampleCount * maxRobotCount * sizeof(float));
	cout << "AllocKdy: " << error << "\n";
	error = hipMalloc((void**)&DevEachDataCount, RegionCount * sizeof(float));
	cout << "AllocEachDataCount: " << error << "\n";
	error = hipMalloc((void**)&DevRobots, 2 * maxRobotCount * sizeof(float));
	cout << "AllocRobots: " << error << "\n";
	for (int i = 0; i < RegionCount; i++)
	{
		error = hipMemcpy(DevDataX + i * maxSampleCount, DataScoreX + i * maxSampleCount, eacheRegionCount[i] * sizeof(float), hipMemcpyHostToDevice);
		cout << "memcpyDataX " << i << ": " << error << "\n";
		error = hipMemcpy(DevDataY + i * maxSampleCount, DataScoreY + i * maxSampleCount, eacheRegionCount[i] * sizeof(float), hipMemcpyHostToDevice);
		cout << "memcpyDataY " << i << ": " << error << "\n";
	}
	error = hipMemcpy(DevEachDataCount, eacheRegionCount, RegionCount * sizeof(float), hipMemcpyHostToDevice);
	cout << "memcpyEachRegionCount: " << error << "\n";
}
void CalculateBallState(GObjectState* robots, int RobotCounts, int N, GObjectState ball, float* Heads, float* Tails, float* TimeHeads, float* TimeTails, int* histo)
{
	if (RobotCounts>0)
	{
		error = hipMemcpy(states, robots, RobotCounts * sizeof(GObjectState), hipMemcpyHostToDevice);
		//cout << "MemcpyState: "<< error << "\n";
		error = hipMemcpy(devHisto2D, zeroArray, 2 * maxRobotCount * sizeof(int) * 2, hipMemcpyHostToDevice);
		//cout << "MemcpyZeroArray: "<< error << "\n";
		dim3 block((threadsPerBlock + AccelSteps) / (AccelSteps + 1), AccelSteps + 1);
		dim3 grid((N + block.x - 1) / block.x, RobotCounts);
		ClaculateTimeKernel << <grid, block >> >(ball, states, RobotCounts, N, _maxRobotSpeed, _ballDecel, _maxRobotAccel, devMinTime2D, 2 * maxRobotCount, maxPoints);
		//	error = hipGetLastError();
		//cout << "CalculateTime: "<< error << "\n";
		//	error = hipDeviceSynchronize();
		//	cout << "Sync: "<< error << "\n";
		grid = dim3((threadsPerBlock + N - 1) / threadsPerBlock, RobotCounts);
		block = dim3(threadsPerBlock);



		MultiObjectKernel << <grid, block >> >(ball, _ballDecel, devMinTime2D, 2 * maxRobotCount, maxPoints, N, devHeads2D, 2 * maxRobotCount, maxLines, devTails2D, 2 * maxRobotCount, maxLines, devtimehead2D, 2 * maxRobotCount, maxLines, devtimetail2D, 2 * maxRobotCount, maxLines, devHisto2D, 2 * maxRobotCount, 2);

		//error = hipGetLastError();
		//cout << "MultiObj: "<< error << "\n";
		//error = hipDeviceSynchronize();
		//	cout << "Sync: "<< error << "\n";

		error = hipMemcpy(pHeads, devHeads2D, 2 * maxRobotCount * maxLines * sizeof(GPosition2D), hipMemcpyDeviceToHost);
		//	cout << "memcpyheads: "<< error << "\n";
		error = hipMemcpy(pTails, devTails2D, 2 * maxRobotCount * maxLines * sizeof(GPosition2D), hipMemcpyDeviceToHost);
		//	cout << "memcpytails: "<< error << "\n";
		error = hipMemcpy(TimeHeads, devtimehead2D, 2 * maxRobotCount * maxLines * sizeof(float), hipMemcpyDeviceToHost);
		//	cout << "memcpytimeheads: "<< error << "\n";
		error = hipMemcpy(TimeTails, devtimetail2D, 2 * maxRobotCount * maxLines * sizeof(float), hipMemcpyDeviceToHost);
		//	cout << "memcpytimetails: "<< error << "\n";
		error = hipMemcpy(histo, devHisto2D, 2 * RobotCounts * sizeof(int), hipMemcpyDeviceToHost);
		//	cout << "memcpyhisto: "<< error << "\n";
		for (int i = 0; i < 2 * maxRobotCount * maxLines; i++)
		{
			Heads[i * 2] = pHeads[i].X;
			Heads[i * 2 + 1] = pHeads[i].Y;
			Tails[i * 2] = pTails[i].X;
			Tails[i * 2 + 1] = pTails[i].Y;
		}
	}
}
void DisposeGamePlanner()
{
	hipFree(devHeads2D);
	hipFree(devTails2D);
	hipFree(devtimehead2D);
	hipFree(devtimetail2D);
	hipFree(devHisto2D);
	hipFree(devMinTime2D);
	hipFree(states);

	hipFree(DevDataX);
	hipFree(DevDataY);
	hipFree(DevPhi);
	hipFree(DevKdx);
	hipFree(DevKdy);
	hipFree(DevEachDataCount);
	free(zeroArray);
}


__device__ GPosition2D  Meet(GPosition2D P, float Ox, float Oy, float obstacleRadi)
{
	float size = (P.X - Ox) * (P.X - Ox) + (P.Y - Oy) * (P.Y - Oy);
	if (size < obstacleRadi * obstacleRadi)
	{
		if (size > 1E-5)
		{
			size = sqrtf(size);
			P.X = (P.X - Ox) * obstacleRadi / size;
			P.Y = (P.Y - Oy) * obstacleRadi / size;
		}
		else
		{
			P.X = Ox + obstacleRadi;
			P.Y = Oy + obstacleRadi;
		}
	}
	return P;
}
__device__ GVector2D MeetCircle(float Ox, float Oy, GVector2D F, float Px, float Py, float R)
{
	float Vx, Vy/*, Vx1, Vy1, Vx2, Vy2*/, tmp/*, tmp2*/;
	/*float f, l, d;
	l = F.X * F.X + F.Y * F.Y;
	f = F.X * Ox - F.X * Px + F.Y * Oy - F.Y * Py;
	if (f <= 0.0)
	d = (Px - Ox)*(Px - Ox) + (Py - Oy)*(Py - Oy);
	else if (f >= l)
	d = (Px + F.X - Ox) * (Px + F.X - Ox) + (Py + F.Y - Oy) * (Py + F.Y - Oy);
	else
	d = (Px + F.X * (f / l) - Ox) * (Px + F.X * (f / l) - Ox) + (Py + F.Y * (f / l) - Oy) * (Py + F.Y * (f / l) - Oy);*/
	Vx = Px + F.X - Ox;
	Vy = Py + F.Y - Oy;
	tmp = Vx * Vx + Vy * Vy;
	if (tmp < R * R)
	{

		if (tmp > 1E-5)
		{
			Vx *= (R / sqrtf(tmp));
			Vy *= (R / sqrtf(tmp));
		}
		else
		{
			Vx = Vy = 0;
		}
		F.X = Ox + Vx - Px;
		F.Y = Oy + Vy - Py;
		/*Vx = Ox - Px;
		Vy = Oy - Py;
		tmp = sqrtf(Vx * Vx + Vy * Vy);
		Vx *= (tmp == 0)? 0: (R / tmp);
		Vy *= (tmp == 0)? 0: (R / tmp);
		tmp = sqrtf(F.X * F.X + F.Y * F.Y) * 0.01;
		if(Vy == 0 && Vx == 0)
		{
		Vx1 = -F.X;
		Vy1 = -F.Y;
		Vx2 = -F.X;
		Vy2 = -F.Y;
		}
		else if(Vy == 0)
		{
		Vx1 = 0;
		Vy1 = tmp ;
		Vx2 = 0;
		Vy2 = -tmp;
		}
		else if(Vx == 0)
		{
		Vx1 = tmp;
		Vy1 = 0;
		Vx2 = -tmp;
		Vy2 = 0;
		}
		else
		{
		tmp2 = tmp / sqrtf(1 + (Vx * Vx)/(Vy * Vy));
		Vx1 = tmp2;
		Vy1 = -(Vx / Vy) * tmp2;
		Vx2 = -tmp2;
		Vy2 = (Vx / Vy) * tmp2;
		}
		if(Vx1 * F.X + Vy1 * F.Y >= 0)
		{
		F.X = Vx1;
		F.Y = Vy1;
		}
		else
		{
		F.X = Vx2;
		F.Y = Vy2;
		}*/
	}
	return F;
}
__global__ void CalculateForcesKernel(float* DevForce, int* DevEachPathCount, int RobotCount, size_t forcePitch, float kSpring, float kSpring2, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = threadIdx.y;
	int k = blockIdx.y;
	int  thisPathCount;
	thisPathCount = DevEachPathCount[k];
	if (i < thisPathCount && j < 2 && k < RobotCount)
	{
		float tmpCurr = tex2D(texB, (float)(2 * i + j), (float)k);
		float tmpNext = tex2D(texB, (float)(2 * (i - 1) + j), (float)k);
		float tmpPrev = tex2D(texB, (float)(2 * (i + 1) + j), (float)k);
		/*	float tmpNnext, tmpNprev;
		if(i  < n)
		tmpNprev = tex2D(texB, (float)j, (float)k);
		else
		tmpNprev = tex2D(texB, (float)2 * (i - n) + j, (float)k);
		if(i >= thisPathCount - n)
		tmpNnext = tex2D(texB, (float)2 * (thisPathCount - 1) + j,(float)k);
		else
		tmpNnext = tex2D(texB, (float)2 * (i + n) + j,(float)k);*/

		if (i == thisPathCount - 1 || i == 0)
		{
			DevForce[k * forcePitch + i * 2 + j] = 0;
		}
		else
			DevForce[k * forcePitch + i * 2 + j] = kSpring * ((tmpNext - tmpCurr) + (tmpPrev - tmpCurr));// + kSpring2 * ((tmpNnext - tmpCurr) + (tmpNprev - tmpCurr)) ;
	}
}
__global__ void ReCalculatePath(float* DevPath, int* DevEachPathCount, int RobotCount, size_t PathPitch, int ObstacleCount)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int k = blockIdx.y;
	int  thisPathCount;

	thisPathCount = DevEachPathCount[k];
	GVector2D F;
	GPosition2D P;
	if (i < thisPathCount && k < RobotCount)
	{
		F = GVector2D(tex2D(texF, (float)2 * i, k), tex2D(texF, (float)2 * i + 1, k));
		P = GPosition2D(tex2D(texB, (float)2 * i, k), tex2D(texB, (float)2 * i + 1, k));
		int ab = tex2D(texV, (float)k, 0);
		int az = tex2D(texV, (float)k, 1);
		int ar = tex2D(texV, (float)k, 3);
		int aoz = tex2D(texV, (float)k, 2);
		for (int j = 0; j < ObstacleCount; j++)
		{
			//O = GPosition2D(tex2D(texC, (float)j, 0), tex2D(texC, (float)j, 1));
			if (i != thisPathCount - 1 && i != 0)
			{
				if (ab == 1 && j == 0)
				{
					F = MeetCircle(tex2D(texC, (float)j, 0), tex2D(texC, (float)j, 1), F, P.X, P.Y, BALL_FORCE);
				}
				else if (az == 1 && j > 0 && j < 4)
				{
					F = MeetCircle(tex2D(texC, (float)j, 0), tex2D(texC, (float)j, 1), F, P.X, P.Y, ZONE_FORCE);
				}
				else if (aoz == 1 && j > 3 && j < 7)
				{
					F = MeetCircle(tex2D(texC, (float)j, 0), tex2D(texC, (float)j, 1), F, P.X, P.Y, OPP_ZONE_FORCE);
				}
				else if (ar == 1 && j > 6 && !(((j - 7) < (RobotCount / 2)) && ((k == (j - 7)) || ((k - (RobotCount / 2)) == (j - 7)))))
				{
					F = MeetCircle(tex2D(texC, (float)j, 0), tex2D(texC, (float)j, 1), F, P.X, P.Y, ROBOT_FORCE);
				}
			}
			/*if(i < thisPathCount - 1 && !(((j - 4) < (RobotCount / 2)) && ((k == (j - 4)) || ((k - (RobotCount / 2)) == (j - 4)))))
			P = Meet(P, tex2D(texC, (float)j, 0), tex2D(texC, (float)j, 1), R );*/
		}
		DevPath[k * PathPitch + 2 * i] = P.X + F.X;
		DevPath[k * PathPitch + 2 * i + 1] = P.Y + F.Y;
	}
}

void ElasticInit(int maxPathCount, int maxRRTCount)
{

	MaxPathCount = maxPathCount;
	MaxRRTCount = maxRRTCount;
	hipChannelFormatDesc channelDescA = hipCreateChannelDesc<float>();
	hipChannelFormatDesc channelDescB = hipCreateChannelDesc<float>();
	hipChannelFormatDesc channelDescF = hipCreateChannelDesc<float>();
	hipChannelFormatDesc channelDescV = hipCreateChannelDesc<float>();

	hipError_t error;
	size_t path_pitch_in_byte, force_pitch_in_byte, obs_pitch_in_byte, avoid_pitch_in_byte;

	error = hipMallocPitch((void**)&DevPath, &path_pitch_in_byte, 2 * MaxPathCount * sizeof(float), MaxRRTCount);
	cout << "PathMalloc: " << error << "\n";
	error = hipMallocPitch((void**)&DevForce, &force_pitch_in_byte, 2 * MaxPathCount * sizeof(float), MaxRRTCount);
	cout << "ForceMalloc: " << error << "\n";
	error = hipMallocPitch((void**)&DevObs, &obs_pitch_in_byte, MAX_OBS_COUNT * sizeof(float), 2);
	cout << "ObsMalloc: " << error << "\n";
	error = hipMallocPitch((void**)&DevAvoid, &avoid_pitch_in_byte, MaxRRTCount * sizeof(float), 4);
	cout << "AvoidMalloc: " << error << "\n";
	error = hipMalloc((void**)&DevEachPathCount, MaxRRTCount * sizeof(int));
	cout << "EachPathCountMalloc: " << error << "\n";


	PathPitch = path_pitch_in_byte / sizeof(float);
	ForcePitch = force_pitch_in_byte / sizeof(float);
	ObsPitch = obs_pitch_in_byte / sizeof(float);
	AvoidPitch = avoid_pitch_in_byte / sizeof(float);

	texB.addressMode[0] = hipAddressModeClamp;
	texB.addressMode[1] = hipAddressModeClamp;
	texB.filterMode = hipFilterModePoint;
	texB.normalized = 0;

	texC.addressMode[0] = hipAddressModeClamp;
	texC.addressMode[1] = hipAddressModeClamp;
	texC.filterMode = hipFilterModePoint;
	texC.normalized = 0;

	texF.addressMode[0] = hipAddressModeClamp;
	texF.addressMode[1] = hipAddressModeClamp;
	texF.filterMode = hipFilterModePoint;
	texF.normalized = 0;

	texV.addressMode[0] = hipAddressModeClamp;
	texV.addressMode[1] = hipAddressModeClamp;
	texV.filterMode = hipFilterModePoint;
	texV.normalized = 0;

	error = hipBindTexture2D(NULL, &texB, DevPath, &channelDescA, MaxPathCount * 2, MaxRRTCount, path_pitch_in_byte);
	cout << "BindPath: " << error << "\n";
	error = hipBindTexture2D(NULL, &texC, DevObs, &channelDescB, MAX_OBS_COUNT, 2, obs_pitch_in_byte);
	cout << "BindObs: " << error << "\n";
	error = hipBindTexture2D(NULL, &texF, DevForce, &channelDescF, MaxPathCount * 2, MaxRRTCount, force_pitch_in_byte);
	cout << "BindForce: " << error << "\n";
	error = hipBindTexture2D(NULL, &texV, DevAvoid, &channelDescV, MaxRRTCount, 4, avoid_pitch_in_byte);
	cout << "BindAvoid: " << error << "\n";
}
void DisposeElastic()
{
	hipUnbindTexture(texB);
	hipUnbindTexture(texC);
	hipUnbindTexture(texF);
	hipUnbindTexture(texV);
	hipFree(DevForce);
	hipFree(DevPath);
	hipFree(DevAvoid);
	hipFree(DevObs);
	hipFree(DevEachPathCount);
}

